#include <hip/hip_runtime.h>

#include <iostream>

#define TILE_SIZE 16  // Assuming each block processes a tile of 16 elements

// Reduction kernel to compute the sum of elements in a tile
__global__ void reduceKernel(int *input, int *output, int N) {
    extern __shared__ float sdata[];

    // Each thread loads a unique element from the input array
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Load the element into shared memory
    if (i < N) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = 0;  // Guard against out-of-bounds access
    }

    // Wait for all threads to load their data
    __syncthreads();

    // Do reduction in shared memory
    for (unsigned int s = TILE_SIZE / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        // Wait for all threads to perform the addition
        __syncthreads();
    }

    // The first thread writes the result to the output array
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

int main() {
    int N = 1024;  // Total number of elements
    int *h_input = new int[N];  // Host input array
    h_input[0]=5;
    h_input[1]=6;
    h_input[2]=7;
    float *h_output = new float[N / TILE_SIZE];  // Host output array for block sums

    // Initialize host input array and allocate device memory
    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, (N / TILE_SIZE) * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    reduceKernel<<<N / TILE_SIZE, TILE_SIZE>>>(d_input, d_output, N);

    // Check for and handle errors here

    // Copy the result back to host memory
    hipMemcpy(h_output, d_output, (N / TILE_SIZE) * sizeof(float), hipMemcpyDeviceToHost);

    // Compute the final result by summing the block sums on the host
    float totalSum = 0;
    for (int i = 0; i < N / TILE_SIZE; ++i) {
        totalSum += h_output[i];
    }

    // Cleanup memory
    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;
    delete[] h_output;

    // Print the result
    printf("Total sum: %f\n", totalSum);

    return 0;
}
