#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;


__device__ void exampleKernel2() {
    // Example values
    float a = 50.0f;
    float b = 100.0f;

    // Compute the maximum of a and b
    float max_value = max(a, b);

    printf("max %f  threadIdxx: %d threadIdxy: %d blockIdxx: %d blockIdxy: %d \n",
          max_value,threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
    // Use max_value for further computation...
}

__global__ void exampleKernel() {
    // Example values
    float a = 5.0f;
    float b = 10.0f;

    // Compute the maximum of a and b
    float max_value = max(a, b);
    printf("blockDim.x: %d blockDim.y: %d gridDim.x: %d gridDim.y: %d \n",
            blockDim.x,blockDim.y,gridDim.x,gridDim.y);
    printf("max %f  threadIdxx: %d threadIdxy: %d blockIdxx: %d blockIdxy: %d \n",
          max_value,threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
    
    //exampleKernel2();
    
    __syncthreads();  // such that the inner loop can use the correct Kj, Vj

    // Use max_value for further computation...
}

int main() {
    dim3 grid_dim(1,1);  // batch_size x num_heads
    dim3 block_dim(2,2);  // Bc threads per block
    const int sram_size =1000;
    exampleKernel<<<grid_dim, block_dim,sram_size>>>();

    hipDeviceSynchronize(); // Ensure the kernel has completed
    return 0;
}
